#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <math.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include "fft_cpu.h"


#define REP 100


double computation_time = 0;
void cmp(double* A, double* B, int length){
   for(int i=0;i<length;i++){
      if(fabs(A[i]-B[i])>1e-6 && fabs((A[i]-B[i])/B[i])>1e-6){
         printf("error in results!\n");
         printf("%f %f\n", A[i], B[i]);
         return;
      }
   }
   printf("results OK!\n");
}

unsigned char* generate_mat(int n, int m) {
   srand(42); // fixed seed
   int size = m*n;
   unsigned char* img = (unsigned char*)malloc(size*sizeof(char));
   if(img==NULL){
      printf("Out of memory! \n");
      exit(-1);
    }
   for(int i=0;i<size;i++){
      img[i] = rand()%256;
   }
   return img;
}
__constant__ int _prime[3] = {2,3,5};
__constant__ double pi = 3.141592653589793;
const double pi_cpu = 3.141592653589793;
constexpr int prime_num = 3;
__global__ void cuda_fft_init(int n, int* radix, int* ex_bit_reversal, Complex* wn_pows){
   int start=0, length = n;
   int r = blockIdx.x * blockDim.x + threadIdx.x;
   int i=r;
   if(r<n){
      for(int t=prime_num-1;t>=0;t--){
         int p = _prime[t];
         int cnt = radix[t];
         for(int _=0;_<cnt;_++){
            int id = i % p;
            i = i/p;
            length /= p;
            start += id*length; 
         }
      }
      ex_bit_reversal[r] = start;
      wn_pows[r] = Complex{cos(2*pi*r/n), -sin(2*pi*r/n)};
   }
}

__forceinline__ __device__ Complex operator *(const Complex a, const Complex b){
    return Complex{a.real*b.real-a.imag*b.imag, a.real*b.imag+a.imag*b.real};
} 

__forceinline__ __device__ Complex operator *(const Complex a, double b){
    return Complex{a.real*b, a.imag*b};
} 

__forceinline__ __device__ Complex operator +(const Complex a, const Complex b){
    return Complex{a.real + b.real, a.imag + b.imag};
} 

__forceinline__ __device__ Complex operator -(const Complex a, const Complex b){
    return Complex{a.real - b.real, a.imag - b.imag};
}


__forceinline__ __device__ Complex wm(int k, int m){
   return Complex{cos(2*k*pi/m), -sin(2*k*pi/m)};
} 


__global__ void fft_cuda_basic_kernel(unsigned char* image, Complex* dft_image, int* ex_bit_reversal, 
             int* radix, Complex* wm_pows, int n, int m)
{
   unsigned int r = blockIdx.x * blockDim.x + threadIdx.x;
   unsigned int threads = blockDim.y;
   unsigned int tid = threadIdx.y;
   if(r < n){
      Complex* dft_line = dft_image + r * m;
      unsigned char* image_line = image + r*m;
      for(int j=tid;j<m;j+=threads) dft_line[j] = Complex{(double)image_line[ex_bit_reversal[j]], 0};
      
      int len=1;
      Complex temp[5], temp2[5];
      __syncthreads();
      for(int t=2;t>=0;t--){
         int cnt = radix[t];
         int p = _prime[t];
         while(cnt--){
            for(int i=len*p*tid;i<m;i+=len*p*threads){
                  for(int j=0;j<len;j++){
                     for(int v=0; v<p; v++) temp[v] = dft_line[i+j+v*len] * wm_pows[m*v*j/(len*p)];
                     for(int v=0; v<p; v++){
                        temp2[v] = temp[0];
                        for(int w=1;w<p;w++) temp2[v] = temp2[v] + wm_pows[(w*v*m/p)%m] * temp[w];
                     }
                     for(int v=0; v<p; v++) dft_line[i+j+v*len] = temp2[v];
                  }
            }
            len *= p;
            __syncthreads();
         }
      }

   }
}

__global__ void fft_cuda_basic_kernel_col(Complex* col_temp, Complex* dft_image, int* ex_bit_reversal, 
             int* radix, Complex* wm_pows, int n, int m)
{
   unsigned int r = blockIdx.x * blockDim.x + threadIdx.x;
   unsigned int threads = blockDim.y;
   unsigned int tid = threadIdx.y;
   if(r < m){
      Complex* dft_col = dft_image + r;
      Complex *col = col_temp + r*n;
      for(int i=tid;i<n;i+=threads) col[i] = dft_col[ex_bit_reversal[i]*m];
      __syncthreads();
      int len=1;
      Complex temp[5], temp2[5];
      for(int t=2;t>=0;t--){
         int cnt = radix[t];
         int p = _prime[t];
         while(cnt--){
            for(int i=len*p*tid;i<n;i+=len*p*threads){
                  for(int j=0;j<len;j++){
                     for(int v=0; v<p; v++) temp[v] = col[i+j+v*len] * wm_pows[n*v*j/(len*p)];
                     for(int v=0; v<p; v++){
                        temp2[v] = temp[0];
                        for(int w=1;w<p;w++) temp2[v] = temp2[v] + wm_pows[(w*v*n/p)%n] * temp[w];
                     }
                     for(int v=0; v<p; v++) col[i+j+v*len] = temp2[v];
                  }
            }
            __syncthreads();
            len *= p;
         }
      }
      for(int i=tid;i<n;i+=threads){
         dft_col[i*m] = col[i];
      }
   }
}


void inline fft2_cuda_basic(unsigned char* image, Complex* dft_image, unsigned char* image_device, Complex* dft_device, int n, int m) {
   int numThreads = 32;
   int numBlocksRow = (n+numThreads-1)/numThreads;
   int numBlocksCol = (m+numThreads-1)/numThreads;
   struct timeval before, after;
   hipMemcpy(image_device, image, m*n*sizeof(unsigned char), hipMemcpyHostToDevice);
   int* radix_device; hipMalloc((void **)&radix_device, 3*sizeof(int));
   int* ex_bit_reversal; hipMalloc((void **)&ex_bit_reversal, max(m,n)*sizeof(int));
   Complex* wm_pows; hipMalloc((void **)&wm_pows, max(m,n)*sizeof(Complex));
   Complex* col_temp; hipMalloc((void **)&col_temp, m*n*sizeof(Complex));
   gettimeofday(&before, NULL);
   int radix[3];
   assert(getradix(m, radix)==1);
   hipMemcpy(radix_device, radix, 3*sizeof(int), hipMemcpyHostToDevice);
   cuda_fft_init<<<numBlocksCol ,numThreads>>>(m, radix_device, ex_bit_reversal, wm_pows);
   fft_cuda_basic_kernel<<<(n+4-1)/4, dim3(4,8)>>>(image_device, dft_device, ex_bit_reversal, radix_device, wm_pows, n, m);

   assert(getradix(n, radix)==1);
   hipMemcpy(radix_device, radix, 3*sizeof(int), hipMemcpyHostToDevice);
   cuda_fft_init<<<numBlocksRow ,numThreads>>>(n, radix_device, ex_bit_reversal, wm_pows);
   fft_cuda_basic_kernel_col<<<(m+4-1)/4, dim3(4,8)>>>(col_temp, dft_device, ex_bit_reversal, radix_device, wm_pows, n, m);


   hipDeviceSynchronize();
   gettimeofday(&after, NULL);
   computation_time += (after.tv_sec + (after.tv_usec / 1000000.0)) -
                      (before.tv_sec + (before.tv_usec / 1000000.0));

   hipMemcpy(dft_image, dft_device, m*n*sizeof(Complex), hipMemcpyDeviceToHost);
   hipFree(radix_device);
   hipFree(ex_bit_reversal);
   hipFree(wm_pows);
   hipFree(col_temp);
}

/*******************************************************************************************************************/
__device__ __forceinline__ void fft1_kernel(Complex *dft_line, Complex* wm_pows, int* radix, int m, int tid, int threads){
   int len=1;
   Complex temp[5], temp2[5];
   for(int t=2;t>=0;t--){
      int cnt = radix[t];
      int p = _prime[t];
      while(cnt--){
         if(len >= threads)
            for(int i=0;i<m;i+=len*p){
               for(int j=tid;j<len;j+=threads){
                  for(int v=0; v<p; v++) temp[v] = dft_line[i+j+v*len] * wm_pows[m*v*j/(len*p)];
                  for(int v=0; v<p; v++){
                     temp2[v] = temp[0];
                     for(int w=1;w<p;w++) temp2[v] = temp2[v] + wm_pows[(w*v*m/p)%m] * temp[w];
                  }
                  for(int v=0; v<p; v++) dft_line[i+j+v*len] = temp2[v];
               }
            }
         else
            for(int i=len*p*tid;i<m;i+=len*p*threads){
                  for(int j=0;j<len;j++){
                     for(int v=0; v<p; v++) temp[v] = dft_line[i+j+v*len] * wm_pows[m*v*j/(len*p)];
                     for(int v=0; v<p; v++){
                        temp2[v] = temp[0];
                        for(int w=1;w<p;w++) temp2[v] = temp2[v] + wm_pows[(w*v*m/p)%m] * temp[w];
                     }
                     for(int v=0; v<p; v++) dft_line[i+j+v*len] = temp2[v];
                  }
            }
         len *= p;
         __syncthreads();
      }
   }
} 

__device__ __forceinline__ void fft1_kernel_unroll(Complex *dft_line, Complex* wm_pows, int* radix, int m, int tid, int threads){
   const Complex w3_1 = Complex{-0.5,-0.8660254037844386}, w3_2 = Complex{-0.5,0.8660254037844386};
   const Complex w5_1 = Complex{0.30901699437494745, -0.9510565162951535},
                     w5_2 = Complex{-0.8090169943749473, -0.5877852522924732},
                     w5_3 = Complex{-0.8090169943749475, 0.587785252292473},
                     w5_4 = Complex{0.30901699437494723, 0.9510565162951536};
   
      
   int len=1;
   Complex temp[5];
   int cnt = radix[2];
   int p = 5;
   while(cnt--){
      if(len >= threads)
         for(int i=0;i<m;i+=len*p){
            for(int j=tid;j<len;j+=threads){
               temp[0] = dft_line[i+j];
               temp[1] = dft_line[i+j+1*len] * wm_pows[m/(len*p)*j];
               temp[2] = dft_line[i+j+2*len] * wm_pows[m/(len*p)*j*2];
               temp[3] = dft_line[i+j+3*len] * wm_pows[m/(len*p)*j*3];
               temp[4] = dft_line[i+j+4*len] * wm_pows[m/(len*p)*j*4];

               dft_line[i+j] = temp[0] + temp[1] + temp[2] + temp[3] + temp[4];
               dft_line[i+j+len] = temp[0] + temp[1]*w5_1 + temp[2]*w5_2 + temp[3]*w5_3 + temp[4]*w5_4;
               dft_line[i+j+2*len] = temp[0] + temp[1]*w5_2 + temp[2]*w5_4 + temp[3]*w5_1 + temp[4]*w5_3;
               dft_line[i+j+3*len] = temp[0] + temp[1]*w5_3 + temp[2]*w5_1 + temp[3]*w5_4 + temp[4]*w5_2;
               dft_line[i+j+4*len] = temp[0] + temp[1]*w5_4 + temp[2]*w5_3 + temp[3]*w5_2 + temp[4]*w5_1;
            }
         }
      else
         for(int i=len*p*tid;i<m;i+=len*p*threads){
            for(int j=0;j<len;j++){
               temp[0] = dft_line[i+j];
               temp[1] = dft_line[i+j+1*len] * wm_pows[m/(len*p)*j];
               temp[2] = dft_line[i+j+2*len] * wm_pows[m/(len*p)*j*2];
               temp[3] = dft_line[i+j+3*len] * wm_pows[m/(len*p)*j*3];
               temp[4] = dft_line[i+j+4*len] * wm_pows[m/(len*p)*j*4];

               dft_line[i+j] = temp[0] + temp[1] + temp[2] + temp[3] + temp[4];
               dft_line[i+j+len] = temp[0] + temp[1]*w5_1 + temp[2]*w5_2 + temp[3]*w5_3 + temp[4]*w5_4;
               dft_line[i+j+2*len] = temp[0] + temp[1]*w5_2 + temp[2]*w5_4 + temp[3]*w5_1 + temp[4]*w5_3;
               dft_line[i+j+3*len] = temp[0] + temp[1]*w5_3 + temp[2]*w5_1 + temp[3]*w5_4 + temp[4]*w5_2;
               dft_line[i+j+4*len] = temp[0] + temp[1]*w5_4 + temp[2]*w5_3 + temp[3]*w5_2 + temp[4]*w5_1;
            }
         }
      len *= p;
      __syncthreads();
   }
   
   cnt = radix[1];
   p = 3;
   while(cnt--){
      if(len >= threads)
         for(int i=0;i<m;i+=len*p){
            for(int j=tid;j<len;j+=threads){
               temp[0] = dft_line[i+j];
               temp[1] = dft_line[i+j+len] * wm_pows[m/(len*p)*j];
               temp[2] = dft_line[i+j+2*len] * wm_pows[m/(len*p)*j*2];
               dft_line[i+j] = temp[0] + temp[1] + temp[2];
               dft_line[i+j+len] = temp[0] + temp[1]*w3_1 + temp[2]*w3_2;
               dft_line[i+j+2*len] = temp[0] + temp[1]*w3_2 + temp[2]*w3_1;
            }
         }
      else
         for(int i=len*p*tid;i<m;i+=len*p*threads){
            for(int j=0;j<len;j++){
               temp[0] = dft_line[i+j];
               temp[1] = dft_line[i+j+len] * wm_pows[m/(len*p)*j];
               temp[2] = dft_line[i+j+2*len] * wm_pows[m/(len*p)*j*2];
               dft_line[i+j] = temp[0] + temp[1] + temp[2];
               dft_line[i+j+len] = temp[0] + temp[1]*w3_1 + temp[2]*w3_2;
               dft_line[i+j+2*len] = temp[0] + temp[1]*w3_2 + temp[2]*w3_1;
            }
         }
      len *= p;
      __syncthreads();
   }
   


   cnt = radix[0];
   p = 2;
   while(cnt--){
      if(len >= threads)
         for(int i=0;i<m;i+=len*p){
            for(int j=tid;j<len;j+=threads){
               temp[0] = dft_line[i+j];
               temp[1] = dft_line[i+j+len] * wm_pows[m/(len*p)*j];
               dft_line[i+j] = temp[0] + temp[1];
               dft_line[i+j+len] = temp[0] - temp[1];
            }
         }
      else
         for(int i=len*p*tid;i<m;i+=len*p*threads){
            for(int j=0;j<len;j++){
               temp[0] = dft_line[i+j];
               temp[1] = dft_line[i+j+len] * wm_pows[m/(len*p)*j];
               dft_line[i+j] = temp[0] + temp[1];
               dft_line[i+j+len] = temp[0] - temp[1];
            }
         }
      len *= p;
      __syncthreads();
   }
} 

__global__ void fft_cuda_kernel(unsigned char* image, Complex* dft_image, int* ex_bit_reversal, 
             int* radix, Complex* wm_pows, int n, int m)
{
   unsigned int r = blockIdx.x;
   unsigned int threads = blockDim.x;
   unsigned int tid = threadIdx.x;


   extern __shared__ Complex dft_line[];
   Complex* dft_res = dft_image + r * m;
   unsigned char* image_line = image + r*m;
   for(int j=tid;j<m;j+=threads) dft_line[j] = Complex{(double)image_line[ex_bit_reversal[j]], 0};
   __syncthreads();
   fft1_kernel(dft_line, wm_pows, radix, m, tid, threads);
   for(int j=tid;j<m;j+=threads) dft_res[j] = dft_line[j];
   
}


__global__ void fft_cuda_kernel_col(Complex* dft_image, int* ex_bit_reversal, 
             int* radix, Complex* wm_pows, int n, int m)
{
   unsigned int r = blockIdx.x;
   unsigned int threads = blockDim.x;
   unsigned int tid = threadIdx.x;

   extern __shared__ Complex col[];
   Complex* dft_col = dft_image + r;
   for(int i=tid;i<n;i+=threads) col[i] = dft_col[ex_bit_reversal[i]*m];
   __syncthreads();
   fft1_kernel(col, wm_pows, radix, n, tid, threads);
   for(int i=tid;i<n;i+=threads){
      dft_col[i*m] = col[i];
   }
}


void inline fft2_cuda(unsigned char* image, Complex* dft_image, unsigned char* image_device, Complex* dft_device, int n, int m) {
   int numThreads = 32;
   int numBlocksRow = (n+numThreads-1)/numThreads;
   int numBlocksCol = (m+numThreads-1)/numThreads;
   struct timeval before, after;
   hipMemcpy(image_device, image, m*n*sizeof(char), hipMemcpyHostToDevice);   int* radix_device; hipMalloc((void **)&radix_device, 3*sizeof(int));
   int* ex_bit_reversal; hipMalloc((void **)&ex_bit_reversal, max(m,n)*sizeof(int));
   Complex* wm_pows; hipMalloc((void **)&wm_pows, max(m,n)*sizeof(Complex));   
   gettimeofday(&before, NULL);
   int radix[3];
   assert(getradix(m, radix)==1);
   hipMemcpy(radix_device, radix, 3*sizeof(int), hipMemcpyHostToDevice);
   cuda_fft_init<<<numBlocksCol ,numThreads>>>(m, radix_device, ex_bit_reversal, wm_pows);   fft_cuda_kernel<<<n, 64, m*sizeof(Complex)>>>(image_device, dft_device, ex_bit_reversal, radix_device, wm_pows, n, m);

   assert(getradix(n, radix)==1);
   hipMemcpy(radix_device, radix, 3*sizeof(int), hipMemcpyHostToDevice);
   cuda_fft_init<<<numBlocksRow ,numThreads>>>(n, radix_device, ex_bit_reversal, wm_pows);
   fft_cuda_kernel_col<<<m, 64, n*sizeof(Complex)>>>(dft_device, ex_bit_reversal, radix_device, wm_pows, n, m);
   hipDeviceSynchronize();
   gettimeofday(&after, NULL);
   computation_time += (after.tv_sec + (after.tv_usec / 1000000.0)) -
                      (before.tv_sec + (before.tv_usec / 1000000.0));   
   hipMemcpy(dft_image, dft_device, m*n*sizeof(Complex), hipMemcpyDeviceToHost);
   hipFree(radix_device);
   hipFree(ex_bit_reversal);
   hipFree(wm_pows);
}

/*********************************************************************************************************/

__global__ void fft_cuda_kernel_unroll(unsigned char* image, Complex* dft_image, int* ex_bit_reversal, 
             int* radix, Complex* wm_pows, int n, int m)
{
   unsigned int r = blockIdx.x;
   unsigned int threads = blockDim.x;
   unsigned int tid = threadIdx.x;


   extern __shared__ Complex dft_line[];
   Complex* dft_res = dft_image + r * m;
   unsigned char* image_line = image + r*m;
   for(int j=tid;j<m;j+=threads) dft_line[j] = Complex{(double)image_line[ex_bit_reversal[j]], 0};
   __syncthreads();
   fft1_kernel_unroll(dft_line, wm_pows, radix, m, tid, threads);
   for(int j=tid;j<m;j+=threads) dft_res[j] = dft_line[j];
   
}

__global__ void fft_cuda_kernel_col_unroll(Complex* dft_image, int* ex_bit_reversal, 
             int* radix, Complex* wm_pows, int n, int m)
{
   unsigned int r = blockIdx.x;
   unsigned int threads = blockDim.x;
   unsigned int tid = threadIdx.x;

   extern __shared__ Complex col[];

   Complex* dft_col = dft_image + r;
   for(int i=tid;i<n;i+=threads) col[i] = dft_col[ex_bit_reversal[i]*m];
   __syncthreads();
   fft1_kernel_unroll(col, wm_pows, radix, n, tid, threads);
   for(int i=tid;i<n;i+=threads){
      dft_col[i*m] = col[i];
   }
}

void inline fft2_cuda_unroll(unsigned char* image, Complex* dft_image, unsigned char* image_device, Complex* dft_device, int n, int m) {
   int numThreads = 32;
   int numBlocksRow = (n+numThreads-1)/numThreads;
   int numBlocksCol = (m+numThreads-1)/numThreads;
   struct timeval before, after;

   hipMemcpy(image_device, image, m*n*sizeof(char), hipMemcpyHostToDevice);
   int* radix_device; hipMalloc((void **)&radix_device, 3*sizeof(int));
   int* ex_bit_reversal; hipMalloc((void **)&ex_bit_reversal, max(m,n)*sizeof(int));
   Complex* wm_pows; hipMalloc((void **)&wm_pows, max(m,n)*sizeof(Complex));
   
   gettimeofday(&before, NULL);
   int radix[3];
   assert(getradix(m, radix)==1);
   hipMemcpy(radix_device, radix, 3*sizeof(int), hipMemcpyHostToDevice);
   cuda_fft_init<<<numBlocksCol ,numThreads>>>(m, radix_device, ex_bit_reversal, wm_pows);
   fft_cuda_kernel_unroll<<<n, 64, m*sizeof(Complex)>>>(image_device, dft_device, ex_bit_reversal, radix_device, wm_pows, n, m);

   assert(getradix(n, radix)==1);
   hipMemcpy(radix_device, radix, 3*sizeof(int), hipMemcpyHostToDevice);
   cuda_fft_init<<<numBlocksRow ,numThreads>>>(n, radix_device, ex_bit_reversal, wm_pows);
   fft_cuda_kernel_col_unroll<<<m, 64, n*sizeof(Complex)>>>(dft_device, ex_bit_reversal, radix_device, wm_pows, n, m);
   hipDeviceSynchronize();
   gettimeofday(&after, NULL);
   computation_time += (after.tv_sec + (after.tv_usec / 1000000.0)) -
                      (before.tv_sec + (before.tv_usec / 1000000.0));

   hipMemcpy(dft_image, dft_device, m*n*sizeof(Complex), hipMemcpyDeviceToHost);
   hipFree(radix_device);hipFree(ex_bit_reversal);hipFree(wm_pows);

}
/*******************************************************************************************************/



constexpr int DEFAULT_M = 1920;
constexpr int DEFAULT_N = 1080;

void inline fft2_cuda_stream(unsigned char* image, Complex* dft_image, unsigned char* image_device, Complex* dft_device, int n, int m) {
   struct timeval before, after;

   const int nstreams = 2;
   hipStream_t streams[nstreams];
   unsigned char* image_device_stream[nstreams];
   Complex* dft_image_stream[nstreams];
   Complex* dft_image_device_stream[nstreams];
   for(int i=0;i<nstreams;i++) {
      hipStreamCreate(&streams[i]);
      hipMalloc((void **)&dft_image_device_stream[i], m*n*sizeof(Complex));
      hipMalloc((void **)&image_device_stream[i], m*n*sizeof(char));
      dft_image_stream[i] = (Complex *)malloc(m*n*sizeof(Complex));
   }

   int numThreads = 32;
   int numBlocksRow = (n+numThreads-1)/numThreads;
   int numBlocksCol = (m+numThreads-1)/numThreads;
   int* radix_device_m; hipMalloc((void **)&radix_device_m, 3*sizeof(int));
   int* ex_bit_reversal_m; hipMalloc((void **)&ex_bit_reversal_m, m*sizeof(int));
   Complex* wm_pows; hipMalloc((void **)&wm_pows, m*sizeof(Complex));
   int* radix_device_n; hipMalloc((void **)&radix_device_n, 3*sizeof(int));
   int* ex_bit_reversal_n; hipMalloc((void **)&ex_bit_reversal_n, n*sizeof(int));
   Complex* wn_pows; hipMalloc((void **)&wn_pows, n*sizeof(Complex));

   int radix[3];
   assert(getradix(m, radix)==1);
   hipMemcpy(radix_device_m, radix, 3*sizeof(int), hipMemcpyHostToDevice);
   cuda_fft_init<<<numBlocksCol ,numThreads>>>(m, radix_device_m, ex_bit_reversal_m, wm_pows);

   assert(getradix(n, radix)==1);
   hipMemcpy(radix_device_n, radix, 3*sizeof(int), hipMemcpyHostToDevice);
   cuda_fft_init<<<numBlocksRow ,numThreads>>>(n, radix_device_n, ex_bit_reversal_n, wn_pows);   
   hipDeviceSynchronize();

gettimeofday(&before, NULL);

   for(int i=0;i<REP;i++){
      int stream_id = i%nstreams;
      hipMemcpyAsync(dft_image_stream[stream_id], dft_image_device_stream[stream_id], m*n*sizeof(Complex), 
                      hipMemcpyDeviceToHost, streams[stream_id]);

      hipMemcpyAsync(image_device_stream[stream_id], image, m*n*sizeof(char), hipMemcpyHostToDevice, streams[stream_id]);
      
      fft_cuda_kernel_unroll<<<n, 64, m*sizeof(Complex), streams[stream_id]>>>(image_device_stream[stream_id], 
                     dft_image_device_stream[stream_id], ex_bit_reversal_m, radix_device_m, wm_pows, n, m);
      fft_cuda_kernel_col_unroll<<<m, 64, n*sizeof(Complex), streams[stream_id]>>>(dft_image_device_stream[stream_id], 
                     ex_bit_reversal_n, radix_device_n, wn_pows, n, m);
   }
   for(int i=0;i<nstreams; i++){
      int stream_id = (REP+i)%nstreams;
      hipMemcpyAsync(dft_image_stream[stream_id], dft_image_device_stream[stream_id], m*n*sizeof(Complex), 
                      hipMemcpyDeviceToHost, streams[stream_id]);
   }
   hipDeviceSynchronize();
   
   gettimeofday(&after, NULL);
   computation_time += (after.tv_sec + (after.tv_usec / 1000000.0)) -
                      (before.tv_sec + (before.tv_usec / 1000000.0));
   
   hipFree(radix_device_m);hipFree(ex_bit_reversal_m);hipFree(wm_pows);
   hipFree(radix_device_n);hipFree(ex_bit_reversal_n);hipFree(wn_pows);
   memcpy(dft_image, dft_image_stream[0], m*n*sizeof(Complex));
   for(int i=0;i<nstreams;i++) {
      free(dft_image_stream[i]);
      hipFree(dft_image_device_stream[i]);
      hipFree(image_device_stream[i]);
   }
   
}

unsigned char frame[DEFAULT_N][DEFAULT_M][3];
int test_stream(){
   int size = DEFAULT_N*DEFAULT_M*3;
   for(int i=0;i<REP;i++){
      for(int i=0;i<size;i++){
         ((char*)(frame))[i] = rand()%256;
      }
   }

   return 0;

}
int main (int argc, char** argv) {
   struct timeval before, after;
   int m, n;
   unsigned char* image = NULL;
   double* GT = NULL;
 
   if (argc > 3) {
      fprintf(stderr, "Usage: %s [martix1] [groundtruthmatrix] \n", argv[0]);
      exit(1);
   }
   else {
      if(argc >= 2){
         image = readImage(argv[1], &n, &m);
      }else{
         n = DEFAULT_N; m = DEFAULT_M;
         image = generate_mat(n, m);
      }
      if(argc==3){
         int _m, _n;
         GT = readGT(argv[2], &_n, &_m);
         if(n!=_n  or m!=_m){
            printf("Size error! \n"); 
            exit(1);
         }
      }
   }

   Complex* dft_image = (Complex *)malloc(m*n*sizeof(Complex));
   Complex* dft_image2 = (Complex *)malloc(m*n*sizeof(Complex));
   if(dft_image == NULL){
      printf("Out of memory! \n");
      exit(-1);
   }
   Complex* dft_device;
   unsigned char* image_device;
   hipMalloc((void **)&dft_device, m*n*sizeof(Complex));
   hipMalloc((void **)&image_device, m*n*sizeof(char));
   unsigned char* image2 = generate_mat(n, m);
   gettimeofday(&before, NULL);
   for(int i=0;i<REP; i++)
     fft2_cuda_unroll(image, dft_image2, image_device, dft_device, n, m);
   gettimeofday(&after, NULL);

   printf("Exec time: %.6f seconds \n", ((after.tv_sec + (after.tv_usec / 1000000.0)) -
               (before.tv_sec + (before.tv_usec / 1000000.0))));
   printf("Computation time: %.6f seconds \n", computation_time);
   free(dft_image);
   free(dft_image2);
   free(image);
   hipFree(image_device);
   hipFree(dft_device);
   return 0;
}

